#include "hip/hip_runtime.h"
#define THREADS 1024

void dot(
  graphblas::Matrix<float>* A,
  graphblas::Matrix<float>* B,
  graphblas::Matrix<float>* C,
  graphblas::Descriptor* desc
)
{
   graphblas::mxm<float,float,float,float>(
       C,
       GrB_NULL,
       GrB_NULL,
       graphblas::PlusMultipliesSemiring<float>(),
       A,
       B,
       desc
   );
}

__global__ void __flatten(int* row_ptr, int* cols, int num_rows) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if(i < num_rows) {
    int start = row_ptr[i];
    int end   = row_ptr[i + 1];
    for(int offset = start; offset < end; offset++) {
      cols[offset] += i * num_rows;
    }
  }
}

void flatten(graphblas::Matrix<float>* A, graphblas::Matrix<float>* flat, bool transpose) {

    int num_edges; A->nvals(&num_edges);
    int num_rows;  A->nrows(&num_rows);

    // Flatten matrix to vector
    int* Av;
    hipMalloc((void**)&Av, num_edges * sizeof(int));
    hipMemcpy(Av, A->matrix_.sparse_.d_csrColInd_, num_edges * sizeof(int), hipMemcpyDeviceToDevice);

    int A_blocks = 1 + (num_rows / THREADS);
    __flatten<<<A_blocks, THREADS>>>(A->matrix_.sparse_.d_csrRowPtr_, Av, num_rows);

    // Convert Av back to GraphBLAS matrix
    int* h_Av = (int*)malloc(num_edges * sizeof(int));
    hipMemcpy(h_Av, Av, num_edges * sizeof(int), hipMemcpyDeviceToHost);

    std::vector<int>   flat_row(num_edges, 0);
    std::vector<int>   flat_col(h_Av, h_Av + num_edges);
    std::vector<float> flat_val(A->matrix_.sparse_.h_csrVal_, A->matrix_.sparse_.h_csrVal_ + num_edges);
    if(!transpose) {
      flat->build(&flat_row, &flat_col, &flat_val, num_edges, GrB_NULL);
    } else {
      flat->build(&flat_col, &flat_row, &flat_val, num_edges, GrB_NULL);
    }
}

float trace(
  graphblas::Matrix<float>* A,
  graphblas::Matrix<float>* B
)
{
    int num_rows;
    A->nrows(&num_rows);

    graphblas::Matrix<float> flat_A(1, num_rows * num_rows);
    flatten(A, &flat_A, false);

    graphblas::Matrix<float> flat_B(num_rows * num_rows, 1);
    flatten(B, &flat_B, true);

    graphblas::Matrix<float> dot_val(1, 1);
    graphblas::Descriptor dot_val_desc;
    dot(&flat_A, &flat_B, &dot_val, &dot_val_desc);

    float * h_trace_val = (float*)malloc(1 * sizeof(float));
    hipMemcpy(h_trace_val, dot_val.matrix_.sparse_.d_csrVal_, 1 * sizeof(float), hipMemcpyDeviceToHost);
    std::cerr << "h_trace_val=" << h_trace_val[0] << std::endl;

    return h_trace_val[0];
}
