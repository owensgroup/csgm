#include "hip/hip_runtime.h"
#define THREADS 1024

// --
// Less verbose wrappers for GraphBLAS functions

void rowsum(
  graphblas::Vector<float>* out,
  graphblas::Matrix<float>* X,
  graphblas::Descriptor* desc
)
{
  graphblas::reduce<float,float,float>(out, GrB_NULL, GrB_NULL, graphblas::PlusMonoid<float>(), X, desc);
}

void easy_mxv(
  graphblas::Vector<float>* out,
  graphblas::Matrix<float>* X,
  graphblas::Vector<float>* y,
  graphblas::Descriptor* desc
)
{
  graphblas::mxv<float, float, float, float>(out, GrB_NULL, GrB_NULL, graphblas::PlusMultipliesSemiring<float>(), X, y, desc);
}

void easy_vxm(
  graphblas::Vector<float>* out,
  graphblas::Vector<float>* y,
  graphblas::Matrix<float>* X,
  graphblas::Descriptor* desc
)
{
  graphblas::vxm<float, float, float, float>(out, GrB_NULL, GrB_NULL, graphblas::PlusMultipliesSemiring<float>(), y, X, desc);
}

void easy_mxm(
  graphblas::Matrix<float>* out,
  graphblas::Matrix<float>* A,
  graphblas::Matrix<float>* B,
  graphblas::Descriptor* desc
)
{
   graphblas::mxm<float,float,float,float>(
       out,
       GrB_NULL,
       GrB_NULL,
       graphblas::PlusMultipliesSemiring<float>(),
       A,
       B,
       desc
   );
}

float sum_reduce(
  float* d_in,
  int num_items
)
{
  float* d_out;
  hipMalloc((void**)&d_out, 1 * sizeof(float));

  void   *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items);
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items);

  float* h_sum = (float*)malloc(sizeof(float));
  hipMemcpy(h_sum, d_out, sizeof(float), hipMemcpyDeviceToHost);
  return *h_sum;
}


// --
// Custome kernels

// __global__ void __flatten(int* row_ptr, int* cols, int n) {
//   int i = threadIdx.x + blockIdx.x * blockDim.x;
//   if(i < n) {
//     int start = row_ptr[i];
//     int end   = row_ptr[i + 1];
//     for(int offset = start; offset < end; offset++) {
//       cols[offset] = cols[offset] + (i * n);
//     }
//   }
// }

// void flatten(graphblas::Matrix<float>* X, graphblas::Matrix<float>* flat, bool transpose) {
//     int num_edges; X->nvals(&num_edges);
//     int num_rows;  X->nrows(&num_rows);

//     // Flatten matrix to vector
//     int* Xv;
//     hipMalloc((void**)&Xv, num_edges * sizeof(int));
//     hipMemcpy(Xv, X->matrix_.sparse_.d_csrColInd_, num_edges * sizeof(int), hipMemcpyDeviceToDevice);

//     int X_blocks = 1 + (num_rows / THREADS);
//     __flatten<<<X_blocks, THREADS>>>(X->matrix_.sparse_.d_csrRowPtr_, Xv, num_rows);

//     // Convert Xv back to GraphBLXS matrix
//     int* h_Xv = (int*)malloc(num_edges * sizeof(int));
//     hipMemcpy(h_Xv, Xv, num_edges * sizeof(int), hipMemcpyDeviceToHost);

//     std::vector<int>   flat_row(num_edges, 0);
//     std::vector<int>   flat_col(h_Xv, h_Xv + num_edges);
//     std::vector<float> flat_val(X->matrix_.sparse_.h_csrVal_, X->matrix_.sparse_.h_csrVal_ + num_edges);
//     if(!transpose) {
//       flat->build(&flat_row, &flat_col, &flat_val, num_edges, GrB_NULL);
//     } else {
//       flat->build(&flat_col, &flat_row, &flat_val, num_edges, GrB_NULL);
//     }
// }



// float cpu_trace(
//   graphblas::Matrix<float>* A,
//   graphblas::Matrix<float>* B,
//   graphblas::Descriptor* desc
// )
// {
//     A->matrix_.sparse_.gpuToCpu();
//     B->matrix_.sparse_.gpuToCpu();

//     int nrows; A->nrows(&nrows);

//     graphblas::Matrix<float> flat_A(1, nrows * nrows); flatten(A, &flat_A, false);
//     hipDeviceSynchronize();

//     graphblas::Matrix<float> flat_B(nrows * nrows, 1); flatten(B, &flat_B, true);
//     hipDeviceSynchronize();

//     graphblas::Matrix<float> trace_mtx(1, 1);
//     easy_mxm(&trace_mtx, &flat_A, &flat_B, desc);
//     trace_mtx.matrix_.sparse_.gpuToCpu();
//     return trace_mtx.matrix_.sparse_.h_csrVal_[0];
// }


__global__ void __flatten2(int* out, int* row_ptr, int* cols, int n) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if(i < n) {
    int start = row_ptr[i];
    int end   = row_ptr[i + 1];
    for(int offset = start; offset < end; offset++) {
      out[offset] = cols[offset] + (i * n);
    }
  }
}


float rowvector_dot(
  float* a_val,
  int*   a_rowptr,
  int*   a_colind,
  float* b_val,
  int*   b_rowptr,
  int*   b_colind,

  int nvals_a,
  int nvals_b,

  int dim
)
{
    hipsparseHandle_t handle = 0;
    hipsparseStatus_t status = hipsparseCreate(&handle);

    // --
    // Transpose B

    int* tb_colind;
    int* tb_rowptr;
    float* tb_val;
    hipMalloc((void**)&tb_colind, nvals_b * sizeof(int));
    hipMalloc((void**)&tb_rowptr, (dim + 1) * sizeof(int));
    hipMalloc((void**)&tb_val, nvals_b * sizeof(float));

    hipsparseScsr2csc(handle, 1, dim, nvals_b,
                     b_val, b_rowptr, b_colind,
                     tb_val, tb_colind, tb_rowptr,
                     HIPSPARSE_ACTION_NUMERIC,
                     HIPSPARSE_INDEX_BASE_ZERO);

    // --
    // Compute dot product

    int* out_row;
    int* out_col;
    float* out_val;
    hipMalloc((void**)&out_row, sizeof(int)*2);
    hipMalloc((void**)&out_col, sizeof(int)*1);
    hipMalloc((void**)&out_val, sizeof(float)*1);

    hipsparseMatDescr_t desc_a;   hipsparseCreateMatDescr(&desc_a);
    hipsparseMatDescr_t desc_b;   hipsparseCreateMatDescr(&desc_b);
    hipsparseMatDescr_t desc_out; hipsparseCreateMatDescr(&desc_out);
    hipsparseScsrgemm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            1, dim, 1,
            desc_a, nvals_a, a_val, a_rowptr, a_colind,
            desc_b, nvals_b, tb_val, tb_rowptr, tb_colind,
            desc_out, out_val, out_row, out_col);

    float* h_out_val = (float*)malloc(1 * sizeof(float));
    hipMemcpy(h_out_val, out_val, 1 * sizeof(float), hipMemcpyDeviceToHost);
    float result = h_out_val[0];

    hipFree(tb_colind);
    hipFree(tb_rowptr);
    hipFree(tb_val);
    hipFree(out_row);
    hipFree(out_col);
    hipFree(out_val);

    return result;
}

void gpu_flatten_matrix(
  int* flat_rowptr,
  int* flat_colind,
  int* rowptr,
  int* colind,
  int nrows,
  int nnz
)
{
    int blocks = 1 + (nrows / THREADS);

    // Flatten columns
    __flatten2<<<blocks, THREADS>>>(flat_colind, rowptr, colind, nrows);

    // Dummy rows
    int * h_flat_rowptr = (int*)malloc(2 * sizeof(int));
    h_flat_rowptr[0] = 0;
    h_flat_rowptr[1] = nnz;
    hipMemcpy(flat_rowptr, h_flat_rowptr, 2 * sizeof(int), hipMemcpyHostToDevice);
}


float gpu_trace(
  graphblas::Matrix<float>* A,
  graphblas::Matrix<float>* B,
  graphblas::Descriptor* desc
)
{
    int nrows; A->nrows(&nrows);
    int blocks = 1 + (nrows / THREADS);

    // --
    // Flatten A

    int A_nnz; A->nvals(&A_nnz);
    int* Af_colind;
    int* Af_rowptr;
    hipMalloc((void**)&Af_colind, A_nnz * sizeof(int));
    hipMalloc((void**)&Af_rowptr, 2 * sizeof(int));
    gpu_flatten_matrix(
      Af_rowptr,
      Af_colind,
      A->matrix_.sparse_.d_csrRowPtr_,
      A->matrix_.sparse_.d_csrColInd_,
      nrows,
      A_nnz
    );

    // --
    // Flatten B

    int B_nnz; B->nvals(&B_nnz);
    int* Bf_colind;
    int* Bf_rowptr;
    hipMalloc((void**)&Bf_colind, B_nnz * sizeof(int));
    hipMalloc((void**)&Bf_rowptr, 2 * sizeof(int));
    gpu_flatten_matrix(
      Bf_rowptr,
      Bf_colind,
      B->matrix_.sparse_.d_csrRowPtr_,
      B->matrix_.sparse_.d_csrColInd_,
      nrows,
      B_nnz
    );


    // --
    // Compute trace

    float trace = rowvector_dot(
      A->matrix_.sparse_.d_csrVal_,
      Af_rowptr,
      Af_colind,

      B->matrix_.sparse_.d_csrVal_, // B will be transposed
      Bf_rowptr,
      Bf_colind,

      A_nnz,
      B_nnz,
      nrows * nrows
    );

    hipFree(Af_colind);
    hipFree(Af_rowptr);
    hipFree(Bf_colind);
    hipFree(Bf_rowptr);

    return trace;
}



// float trace(
//   graphblas::Matrix<float>* A,
//   graphblas::Matrix<float>* B,
//   graphblas::Descriptor* desc
// )
// {
//     A->matrix_.sparse_.gpuToCpu();
//     B->matrix_.sparse_.gpuToCpu();

//     int nrows; A->nrows(&nrows);

//     graphblas::Matrix<float> flat_A(1, nrows * nrows); flatten(A, &flat_A);
//     hipDeviceSynchronize();

//     graphblas::Matrix<float> flat_B(1, nrows * nrows); flatten(B, &flat_B);
//     hipDeviceSynchronize();

//     hipsparseHandle_t handle = 0;
//     hipsparseStatus_t status = hipsparseCreate(&handle);
//     hipsparseMatDescr_t desc_a;   hipsparseCreateMatDescr(&desc_a);
//     hipsparseMatDescr_t desc_b;   hipsparseCreateMatDescr(&desc_b);
//     hipsparseMatDescr_t desc_out; hipsparseCreateMatDescr(&desc_out);

//     int baseC, nnzC;
//     int* csrRowPtrC;
//     int* csrColIndC;
//     float* csrValC;

//     int nvals_a; flat_A.nvals(&nvals_a);

//     int * a_row;
//     int * a_col;
//     float * a_val;
//     hipMalloc((void**)&a_row, sizeof(int) * 2);
//     hipMalloc((void**)&a_col, sizeof(int) * nvals_a);
//     hipMalloc((void**)&a_val, sizeof(float) * nvals_a);
//     hipMemcpy(a_row, flat_A.matrix_.sparse_.d_csrRowPtr_, sizeof(int)   * 2, hipMemcpyDeviceToDevice);
//     hipMemcpy(a_col, flat_A.matrix_.sparse_.d_csrColInd_, sizeof(int)   * nvals_a, hipMemcpyDeviceToDevice);
//     hipMemcpy(a_val, flat_A.matrix_.sparse_.d_csrVal_,    sizeof(float) * nvals_a, hipMemcpyDeviceToDevice);

//     int * h_a_row = (int*)malloc(sizeof(int) * 2);
//     int * h_a_col = (int*)malloc(sizeof(int) * nvals_a);
//     float * h_a_val = (float*)malloc(sizeof(float) * nvals_a);
//     hipMemcpy(h_a_row, a_row, sizeof(int) * 2, hipMemcpyDeviceToHost);
//     hipMemcpy(h_a_col, a_col, sizeof(int) * nvals_a, hipMemcpyDeviceToHost);
//     hipMemcpy(h_a_val, a_val, sizeof(float) * nvals_a, hipMemcpyDeviceToHost);

//     std::cerr << "nvals_a=" << nvals_a << std::endl;
//     for(int i = 0; i < 2; i ++) {
//       std::cerr << i << " " << h_a_row[i] << std::endl;
//     }
//     for(int i = 0; i < 10; i ++) {
//       std::cerr << i << " " << h_a_col[i] << " " << h_a_val[i] << std::endl;
//     }

//     // --

//     int nvals_b; flat_B.nvals(&nvals_b);

//     int * b_row;
//     int * b_col;
//     float * b_val;
//     hipMalloc((void**)&b_row, sizeof(int) * 2);
//     hipMalloc((void**)&b_col, sizeof(int) * nvals_b);
//     hipMalloc((void**)&b_val, sizeof(float) * nvals_b);
//     hipMemcpy(b_row, flat_B.matrix_.sparse_.d_csrRowPtr_, sizeof(int)   * 2, hipMemcpyDeviceToDevice);
//     hipMemcpy(b_col, flat_B.matrix_.sparse_.d_csrColInd_, sizeof(int)   * nvals_b, hipMemcpyDeviceToDevice);
//     hipMemcpy(b_val, flat_B.matrix_.sparse_.d_csrVal_,    sizeof(float) * nvals_b, hipMemcpyDeviceToDevice);

//     int * h_b_row = (int*)malloc(sizeof(int) * 2);
//     int * h_b_col = (int*)malloc(sizeof(int) * nvals_b);
//     float * h_b_val = (float*)malloc(sizeof(float) * nvals_b);
//     hipMemcpy(h_b_row, b_row, sizeof(int) * 2, hipMemcpyDeviceToHost);
//     hipMemcpy(h_b_col, b_col, sizeof(int) * nvals_b, hipMemcpyDeviceToHost);
//     hipMemcpy(h_b_val, b_val, sizeof(float) * nvals_b, hipMemcpyDeviceToHost);

//     std::cerr << "nvals_b=" << nvals_b << std::endl;
//     for(int i = 0; i < 2; i ++) {
//       std::cerr << i << " " << h_b_row[i] << std::endl;
//     }
//     for(int i = 0; i < 10; i ++) {
//       std::cerr << i << " " << h_b_col[i] << " " << h_b_val[i] << std::endl;
//     }

//     // nnzTotalDevHostPtr points to host memory
//     int *nnzTotalDevHostPtr = &nnzC;
//     hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);
//     hipMalloc((void**)&csrRowPtrC, sizeof(int) * 2);
//     hipsparseXcsrgemmNnz(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
//             1, nrows * nrows, 1,
//             desc_a, nvals_a, a_row, a_col,
//             desc_b, nvals_b, b_row, b_col,
//             desc_out, csrRowPtrC, nnzTotalDevHostPtr );

//     // // hipDeviceSynchronize();
//     // // if (NULL != nnzTotalDevHostPtr){
//     // //     nnzC = *nnzTotalDevHostPtr;
//     // // }else{
//     // //     hipMemcpy(&nnzC, csrRowPtrC + 1, sizeof(int), hipMemcpyDeviceToHost);
//     // //     hipMemcpy(&baseC, csrRowPtrC, sizeof(int), hipMemcpyDeviceToHost);
//     // //     nnzC -= baseC;
//     // // }
//     // // hipDeviceSynchronize();
//     // // hipMalloc((void**)&csrColIndC, sizeof(int)*nnzC);
//     // // hipMalloc((void**)&csrValC, sizeof(float)*nnzC);
//     // // hipDeviceSynchronize();
//     // // hipsparseScsrgemm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
//     // //         1, nrows * nrows, 1,
//     // //         desc_a, nvals_a, a_val, a_row, a_col,
//     // //         desc_b, nvals_b, b_val, b_row, b_col,
//     // //         desc_out, csrValC, csrRowPtrC, csrColIndC);
//     // // hipDeviceSynchronize();

//     std::cerr << "returning" << std::endl;

//     return -1;

//     // graphblas::Matrix<float> trace_mtx(1, 1);
//     // easy_mxm(&trace_mtx, &flat_A, &flat_B, desc);
//     // trace_mtx.matrix_.sparse_.gpuToCpu();
//     // return trace_mtx.matrix_.sparse_.h_csrVal_[0];
// }



void add_matrix(
    graphblas::Matrix<float>* A,
    graphblas::Matrix<float>* B,
    graphblas::Matrix<float>* C,
    const float alpha,
    const float beta
)
{
  // This copies data GPU -> CPU -> GPU
  // ATTN CARL

  hipsparseHandle_t handle = 0;
  hipsparseStatus_t status = hipsparseCreate(&handle);

  int nrows; A->nrows(&nrows);
  int ncols; A->ncols(&ncols);
  int nvals_a; A->nvals(&nvals_a);
  int nvals_b; B->nvals(&nvals_b);

  hipsparseMatDescr_t desc_a;   hipsparseCreateMatDescr(&desc_a);
  hipsparseMatDescr_t desc_b;   hipsparseCreateMatDescr(&desc_b);
  hipsparseMatDescr_t desc_out; hipsparseCreateMatDescr(&desc_out);

  int minval_out, nvals_out;
  int* d_indptr_out;
  int* d_indices_out;
  float* d_row_out;

  // nnzTotalDevHostPtr points to host memory
  int *nnzTotalDevHostPtr = &nvals_out;
  hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);
  hipMalloc((void**)&d_indptr_out, sizeof(int) * (nrows + 1));
  hipsparseXcsrgeamNnz(
    handle, nrows, ncols,
    desc_a, nvals_a, A->matrix_.sparse_.d_csrRowPtr_, A->matrix_.sparse_.d_csrColInd_,
    desc_b, nvals_b, B->matrix_.sparse_.d_csrRowPtr_, B->matrix_.sparse_.d_csrColInd_,
    desc_out, d_indptr_out, nnzTotalDevHostPtr
  );

  if (NULL != nnzTotalDevHostPtr){
      nvals_out = *nnzTotalDevHostPtr;
  }else{
      hipMemcpy(&nvals_out, d_indptr_out + nrows, sizeof(int), hipMemcpyDeviceToHost);
      hipMemcpy(&minval_out, d_indptr_out, sizeof(int), hipMemcpyDeviceToHost);
      nvals_out -= minval_out;
  }
  hipMalloc((void**)&d_indices_out, sizeof(int)   * nvals_out);
  hipMalloc((void**)&d_row_out,     sizeof(float) * nvals_out);
  hipsparseScsrgeam(
    handle, nrows, ncols,

    &alpha,
    desc_a, nvals_a, A->matrix_.sparse_.d_csrVal_, A->matrix_.sparse_.d_csrRowPtr_, A->matrix_.sparse_.d_csrColInd_,

    &beta,
    desc_b, nvals_b, B->matrix_.sparse_.d_csrVal_, B->matrix_.sparse_.d_csrRowPtr_, B->matrix_.sparse_.d_csrColInd_,

    desc_out, d_row_out, d_indptr_out, d_indices_out
  );

  float * h_val     = (float*)malloc(sizeof(float) * nvals_out);
  int   * h_indices = (int*)malloc(sizeof(int)     * nvals_out);
  int   * h_indptr  = (int*)malloc(sizeof(int)     * (nrows + 1));

  hipMemcpy(h_val,     d_row_out,     sizeof(float) * nvals_out,   hipMemcpyDeviceToHost);
  hipMemcpy(h_indices, d_indices_out, sizeof(int)   * nvals_out,   hipMemcpyDeviceToHost);
  hipMemcpy(h_indptr,  d_indptr_out,  sizeof(int)   * (nrows + 1), hipMemcpyDeviceToHost);

  std::cerr << "nvals_out=" << nvals_out << std::endl;
  std::vector<int> vec_row;
  for(int i = 0; i < nrows; i++) {
    int start = h_indptr[i];
    int end   = h_indptr[i + 1];
    for(int offset = start; offset < end; offset++) {
      vec_row.push_back(i);
    }
  }
  std::vector<int>   vec_col(h_indices, h_indices + nvals_out);
  std::vector<float> vec_val(h_val,     h_val + nvals_out);
  C->build(&vec_row, &vec_col, &vec_val, nvals_out, GrB_NULL);
  C->print();
}