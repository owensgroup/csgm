#include "hip/hip_runtime.h"
// auction.cu

#ifndef MAIN_AUCTION
#define MAIN_AUCTION

#include <iostream>
#include <string>
#include <fstream>
#include <stdio.h>
#include <stdlib.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// --
// Define constants


#define __RUN_VARS
#define MAX_NODES       20000 // Dimension of problem
#define BLOCKSIZE       32    // How best to set this?
#define AUCTION_MAX_EPS 1.0   // Larger values mean solution is more approximate
#define AUCTION_MIN_EPS 1.0
#define AUCTION_FACTOR  0.0
#define NUM_RUNS        10

#include "auction_kernel_csr.cu"

int run_auction(
    int    num_nodes,
    int    num_edges,

    float* d_data,      // data
    int*   d_offsets,   // offsets for items
    int*   d_columns,

    int*   d_person2item, // results

    float auction_max_eps,
    float auction_min_eps,
    float auction_factor,

    int num_runs,
    int verbose
)
{
    float* h_data    = (float*)malloc(num_edges * sizeof(float));
    int*   h_offsets = (int*)malloc((num_nodes + 1) * sizeof(int));
    int*   h_columns = (int*)malloc(num_edges * sizeof(int));

    // --
    // CUDA options

    dim3 dimBlock(BLOCKSIZE, 1, 1);
    int gx = ceil(num_nodes / (double) dimBlock.x);
    dim3 dimGrid(gx, 1, 1);

    // --
    // Declare variables

    int* d_item2person;

    float* d_bids;
    float* d_prices;
    int*   d_bidders;
    int*   d_sbids;
    int    h_numAssign;
    int*   d_numAssign;
    float* d_rand;

    // --
    // Allocate device memory

    hipMalloc((void **)&d_numAssign,   1                     * sizeof(int)) ;
    hipMalloc((void **)&d_item2person, num_nodes             * sizeof(int));
    hipMalloc((void **)&d_prices,      num_nodes             * sizeof(float));
    hipMalloc((void **)&d_sbids,       num_nodes             * sizeof(int));
    hipMalloc((void **)&d_bids,        num_nodes * num_nodes * sizeof(float));
    hipMalloc((void **)&d_bidders,     num_nodes * num_nodes * sizeof(int)); // unused
    hipMalloc((void **)&d_rand,        num_nodes * num_nodes * sizeof(float)) ;

    // --
    // Copy from host to device

    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 123);
    hiprandGenerateUniform(gen, d_rand, num_nodes * num_nodes);

    for(int run_num = 0; run_num < num_runs; run_num++) {
        hipMemset(d_prices, 0.0, num_nodes * sizeof(float));

        // Start timer
        hipEvent_t start, stop;
        float milliseconds = 0;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        float auction_eps = auction_max_eps;
        while(auction_eps >= auction_min_eps) {
            h_numAssign = 0;
            hipMemset(d_bidders,        0, num_nodes * num_nodes * sizeof(int)); // unused
            hipMemset(d_person2item,   -1, num_nodes * sizeof(int));
            hipMemset(d_item2person,   -1, num_nodes * sizeof(int));
            hipMemset(d_numAssign,      0, 1         * sizeof(int));
            hipDeviceSynchronize();

            int counter = 0;
            while(h_numAssign < num_nodes){
                counter += 1;
                hipMemset(d_bids,  0, num_nodes * num_nodes * sizeof(float));
                hipMemset(d_sbids, 0, num_nodes * sizeof(int));
                hipDeviceSynchronize();

                run_bidding<<<dimBlock, dimGrid>>>(
                    num_nodes,

                    d_data,
                    d_offsets,
                    d_columns,

                    d_person2item,
                    d_bids,
                    d_bidders,
                    d_sbids,
                    d_prices,
                    auction_eps,
                    d_rand
                );
                run_assignment<<<dimBlock, dimGrid>>>(
                    num_nodes,
                    d_person2item,
                    d_item2person,
                    d_bids,
                    d_bidders,
                    d_sbids,
                    d_prices,
                    d_numAssign
                );
                hipDeviceSynchronize();

                hipMemcpy(&h_numAssign, d_numAssign, sizeof(int) * 1, hipMemcpyDeviceToHost);
                // std::cerr << "h_numAssign=" << h_numAssign << std::endl;
            }
            if(verbose) {
                std::cerr << "counter=" << counter << std::endl;
            }

            auction_eps *= auction_factor;
        }
        hipDeviceSynchronize();

        // Stop timer
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        if(verbose) {
            std::cerr <<
                "run_num="         << run_num      <<
                " | h_numAssign="  << h_numAssign  <<
                " | milliseconds=" << milliseconds << std::endl;
        }

        hipDeviceSynchronize();
     }

    hipFree(d_item2person);
    hipFree(d_bids);
    hipFree(d_prices);
    hipFree(d_bidders);
    hipFree(d_sbids);
    hipFree(d_numAssign);
    hipFree(d_rand);

    return 0;
} // end run_auction

#endif
