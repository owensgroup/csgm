#include "hip/hip_runtime.h"
// auction.cu

#ifndef MAIN_AUCTION
#define MAIN_AUCTION

#include <iostream>
#include <string>
#include <fstream>
#include <stdio.h>
#include <stdlib.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// --
// Define constants


#define __RUN_VARS
#define MAX_NODES       20000 // Dimension of problem
#define BLOCKSIZE       32    // How best to set this?
#define AUCTION_MAX_EPS 1.0   // Larger values mean solution is more approximate
#define AUCTION_MIN_EPS 1.0
#define AUCTION_FACTOR  0.0
#define NUM_RUNS        10

#include "auction_kernel_csr.cu"


int load_data(float *raw_data) {
    std::ifstream input_file("graph", std::ios_base::in);

    std::cerr << "load_data: start" << std::endl;
    int i = 0;
    float val;
    while(input_file >> val) {
        raw_data[i] = val;
        i++;
        if(i > MAX_NODES * MAX_NODES) {
            std::cerr << "load_data: ERROR -- data file too large" << std::endl;
            return -1;
        }
    }
    std::cerr << "load_data: finish" << std::endl;
    return (int)sqrt(i);
}


int run_auction(
    int    num_nodes,
    int    num_edges,

    float* h_data,      // data
    int*   h_offsets,   // offsets for items
    int*   h_columns,

    int*   h_person2item, // results

    float auction_max_eps,
    float auction_min_eps,
    float auction_factor,

    int num_runs,
    int verbose
)
{
    // --
    // CUDA options

    dim3 dimBlock(BLOCKSIZE, 1, 1);
    int gx = ceil(num_nodes / (double) dimBlock.x);
    dim3 dimGrid(gx, 1, 1);

    // --
    // Declare variables

    float* d_data;
    int*   d_offsets;
    int*   d_columns;

    int* d_person2item;
    int* d_item2person;

    float* d_bids;
    float* d_prices;
    int*   d_bidders; // unused
    int*   d_sbids;

    int  h_numAssign;
    int* d_numAssign = 0;
    float* d_rand;

    // --
    // Allocate device memory
    hipMalloc((void **)&d_data,    num_edges * sizeof(float));
    hipMalloc((void **)&d_columns, num_edges * sizeof(float));
    hipMalloc((void **)&d_offsets, (num_nodes + 1) * sizeof(int));

    hipMalloc((void **)&d_person2item, num_nodes * sizeof(int));
    hipMalloc((void **)&d_item2person, num_nodes * sizeof(int));

    hipMalloc((void **)&d_bids,    num_nodes * num_nodes * sizeof(float));
    hipMalloc((void **)&d_prices,  num_nodes * sizeof(float));
    hipMalloc((void **)&d_bidders, num_nodes * num_nodes * sizeof(int)); // unused
    hipMalloc((void **)&d_sbids,   num_nodes * sizeof(int));

    hipMalloc((void **)&d_numAssign, 1 * sizeof(int)) ;
    hipMalloc((void **)&d_rand,      num_nodes * num_nodes * sizeof(float)) ;

    // --
    // Copy from host to device

    hipMemcpy(d_data,    h_data,    num_edges       * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_columns, h_columns, num_edges       * sizeof(int),   hipMemcpyHostToDevice);
    hipMemcpy(d_offsets, h_offsets, (num_nodes + 1) * sizeof(int),   hipMemcpyHostToDevice);

    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 123);
    hiprandGenerateUniform(gen, d_rand, num_nodes * num_nodes);

    for(int run_num = 0; run_num < num_runs; run_num++) {

        hipMemset(d_prices, 0.0, num_nodes * sizeof(float));

        // Start timer
        hipEvent_t start, stop;
        float milliseconds = 0;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        float auction_eps = auction_max_eps;
        while(auction_eps >= auction_min_eps) {
            h_numAssign = 0;
            hipMemset(d_bidders,        0, num_nodes * num_nodes * sizeof(int)); // unused
            hipMemset(d_person2item,   -1, num_nodes * sizeof(int));
            hipMemset(d_item2person,   -1, num_nodes * sizeof(int));
            hipMemset(d_numAssign,      0, 1         * sizeof(int));
            hipDeviceSynchronize();

            int counter = 0;
            while(h_numAssign < num_nodes){
                counter += 1;
                hipMemset(d_bids,  0, num_nodes * num_nodes * sizeof(float));
                hipMemset(d_sbids, 0, num_nodes * sizeof(int));
                hipDeviceSynchronize();

                run_bidding<<<dimBlock, dimGrid>>>(
                    num_nodes,

                    d_data,
                    d_offsets,
                    d_columns,

                    d_person2item,
                    d_bids,
                    d_bidders,
                    d_sbids,
                    d_prices,
                    auction_eps,
                    d_rand
                );
                run_assignment<<<dimBlock, dimGrid>>>(
                    num_nodes,
                    d_person2item,
                    d_item2person,
                    d_bids,
                    d_bidders,
                    d_sbids,
                    d_prices,
                    d_numAssign
                );
                hipDeviceSynchronize();

                hipMemcpy(&h_numAssign, d_numAssign, sizeof(int) * 1, hipMemcpyDeviceToHost);
                // std::cerr << "h_numAssign=" << h_numAssign << std::endl;
            }
            if(verbose) {
                std::cerr << "counter=" << counter << std::endl;
            }

            auction_eps *= auction_factor;
        }
        hipDeviceSynchronize();

        // Stop timer
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        if(verbose) {
            std::cerr <<
                "run_num="         << run_num      <<
                " | h_numAssign="  << h_numAssign  <<
                " | milliseconds=" << milliseconds << std::endl;
        }

        hipDeviceSynchronize();
     }

    // Read out results
    hipMemcpy(h_person2item, d_person2item, sizeof(int) * num_nodes, hipMemcpyDeviceToHost);

    hipFree(d_data);
    hipFree(d_columns);
    hipFree(d_offsets);
    hipFree(d_person2item);
    hipFree(d_item2person);
    hipFree(d_bids);
    hipFree(d_prices);
    hipFree(d_bidders);
    hipFree(d_sbids);
    hipFree(d_numAssign);
    hipFree(d_rand);

    return 0;
} // end run_auction

#endif
