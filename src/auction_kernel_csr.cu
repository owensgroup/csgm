
#include <hip/hip_runtime.h>
#ifndef __AUCTION_VARS
#define EMPTY_COL -99
#define BIG_NEGATIVE -999999
#endif

__global__ void run_bidding(
    const int num_nodes,

    float *data,
    int *offsets,
    int *columns,

    int *person2item,
    float *bids,
    int *sbids,
    float *prices,
    float auction_eps,

    float *rand
)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < num_nodes){
        if(person2item[i] == -1) {

            int start_idx = offsets[i];
            int end_idx   = offsets[i + 1];
            // printf("i=%d | num_entries=%d\n", i, end_idx - start_idx);
            // if(start_idx == end_idx) return;

            int top1_col;
            float top1_val = BIG_NEGATIVE;
            float top2_val = BIG_NEGATIVE;

            int col;
            float tmp_val;

            // Check all of the zero entries
            for(int col = 0; col < num_nodes; col++) {
                tmp_val = -prices[col];
                if(tmp_val >= top1_val) {
                    if(
                        (tmp_val > top1_val) // ||
                        // (rand[i * num_nodes + col] >= rand[i * num_nodes + top1_col]) // tiebreaker
                    ) {
                        top2_val = top1_val;
                        top1_col = col;
                        top1_val = tmp_val;
                    }
                } else if(tmp_val > top2_val) {
                    top2_val = tmp_val;
                }
            }

            // Check nonzero entries
            for(int idx = start_idx; idx < end_idx; idx++){
                col = columns[idx];
                if(col == EMPTY_COL) break;
                tmp_val = data[idx] - prices[col];

                if(tmp_val >= top1_val) {
                    if(
                        (tmp_val > top1_val) // ||
                        // (rand[i * num_nodes + col] >= rand[i * num_nodes + top1_col]) // tiebreaker
                    ) {
                        top2_val = top1_val;
                        top1_col = col;
                        top1_val = tmp_val;
                    }
                } else if(tmp_val > top2_val) {
                    top2_val = tmp_val;
                }
            }

            // if(top2_val == BIG_NEGATIVE) {

            // }

            float bid = top1_val - top2_val + auction_eps;
            bids[num_nodes * top1_col + i] = bid;
            atomicMax(sbids + top1_col, 1);
        }
    }
}


__global__ void run_assignment(
    const int num_nodes,
    int *person2item,
    int *item2person,
    float *bids,
    int *sbids,
    float *prices,
    int *num_assigned
)
{

    int j = blockDim.x * blockIdx.x + threadIdx.x;
    if(j < num_nodes) {
        if(sbids[j] != 0) {
            float high_bid  = -1;
            int high_bidder = -1;

            float tmp_bid;
            for(int i = 0; i < num_nodes; i++){
                tmp_bid = bids[num_nodes * j + i];
                if(tmp_bid > high_bid){
                    high_bid    = tmp_bid;
                    high_bidder = i;
                }
            }

            int current_person = item2person[j];
            if(current_person != -1){
                person2item[current_person] = -1;
            } else {
                atomicAdd(num_assigned, 1);
            }

            prices[j]                += high_bid;
            person2item[high_bidder] = j;
            item2person[j]           = high_bidder;
        }
    }
}
