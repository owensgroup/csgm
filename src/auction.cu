// auction.cu

#ifndef MAIN_AUCTION
#define MAIN_AUCTION

#include <iostream>
#include <string>
#include <fstream>
#include <stdio.h>
#include <stdlib.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "auction_kernel_csr.cu"

#define THREADS 8

int run_auction(
    int    num_nodes,
    int    num_edges,

    float* d_data,      // data
    int*   d_offsets,   // offsets for items
    int*   d_columns,

    int*   d_person2item, // results

    float auction_max_eps,
    float auction_min_eps,
    float auction_factor,

    int num_runs,
    int verbose
)
{
    // float* h_data    = (float*)malloc(num_edges * sizeof(float));
    // int*   h_offsets = (int*)malloc((num_nodes + 1) * sizeof(int));
    // int*   h_columns = (int*)malloc(num_edges * sizeof(int));
    // std::cerr << "num_edges=" << num_edges << std::endl;
    // hipMemcpy(h_columns, d_columns, num_edges * sizeof(int), hipMemcpyDeviceToHost);
    // hipMemcpy(h_data, d_data, num_edges * sizeof(float), hipMemcpyDeviceToHost);
    // for(int i = 0; i < 20; i++) {
    //     std::cerr << "h_columns[" << i << "]=" << h_columns[i] << std::endl;
    //     std::cerr << "   h_data[" << i << "]=" << h_data[i] << std::endl;
    // }

    int block = 1 + num_nodes / THREADS;

    // --
    // Declare variables

    int* d_item2person;

    float* d_bids;
    float* d_prices;
    int*   d_sbids;
    int    h_numAssign;
    int*   d_numAssign;
    float* d_rand;

    // --
    // Allocate device memory

    hipMalloc((void **)&d_numAssign,   1                     * sizeof(int)) ;
    hipMalloc((void **)&d_item2person, num_nodes             * sizeof(int));
    hipMalloc((void **)&d_prices,      num_nodes             * sizeof(float));
    hipMalloc((void **)&d_sbids,       num_nodes             * sizeof(int));
    hipMalloc((void **)&d_bids,        num_nodes * num_nodes * sizeof(float));
    hipMalloc((void **)&d_rand,        num_nodes * num_nodes * sizeof(float));

    // --
    // Copy from host to device

    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 123);
    hiprandGenerateUniform(gen, d_rand, num_nodes * num_nodes);

    for(int run_num = 0; run_num < num_runs; run_num++) {
        hipMemset(d_prices, 0.0, num_nodes * sizeof(float));

        // Start timer
        hipEvent_t start, stop;
        float milliseconds = 0;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        float auction_eps = auction_max_eps;
        while(auction_eps >= auction_min_eps) {
            h_numAssign = 0;
            hipMemset(d_person2item,   -1, num_nodes * sizeof(int));
            hipMemset(d_item2person,   -1, num_nodes * sizeof(int));
            hipMemset(d_numAssign,      0, 1         * sizeof(int));
            hipDeviceSynchronize();

            int counter = 0;
            while(h_numAssign < num_nodes){
                counter += 1;
                hipMemset(d_bids,  0, num_nodes * num_nodes * sizeof(float));
                hipMemset(d_sbids, 0, num_nodes * sizeof(int));

                run_bidding<<<block, THREADS>>>(
                    num_nodes,

                    d_data,
                    d_offsets,
                    d_columns,

                    d_person2item,
                    d_bids,
                    d_sbids,
                    d_prices,
                    auction_eps,
                    d_rand
                );
                run_assignment<<<block, THREADS>>>(
                    num_nodes,
                    d_person2item,
                    d_item2person,
                    d_bids,
                    d_sbids,
                    d_prices,
                    d_numAssign
                );

                hipMemcpy(&h_numAssign, d_numAssign, sizeof(int) * 1, hipMemcpyDeviceToHost);
            }
            if(verbose) {
                std::cerr << "counter=" << counter << std::endl;
            }

            auction_eps *= auction_factor;
        }
        hipDeviceSynchronize();

        // Stop timer
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        if(verbose) {
            std::cerr <<
                "run_num="         << run_num      <<
                " | h_numAssign="  << h_numAssign  <<
                " | milliseconds=" << milliseconds << std::endl;
        }

        hipDeviceSynchronize();
     }

    hipFree(d_item2person);
    hipFree(d_bids);
    hipFree(d_prices);
    hipFree(d_sbids);
    hipFree(d_numAssign);
    hipFree(d_rand);

    return 0;
} // end run_auction

#endif
